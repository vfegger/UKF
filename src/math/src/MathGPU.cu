#include "../include/MathGPU.hpp"

void MathGPU::InitializeGPUContext(int device)
{
    hipError_t status = hipSetDevice(device);
    if (status != hipSuccess)
    {
        hipGetDeviceProperties(MathGPU::properties, device);
    }
    else
    {
        std::cout << "Cuda-enabled context was not able to launch properly with the chosen device.\n";
    }
}

void MathGPU::PrintMatrix(double *matrix_in, unsigned lengthX_in, unsigned lengthY_in, unsigned precision)
{
    // Get values from GPU pointer to CPU pointer
    double *matrix_aux = new double[lengthX_in * lengthY_in];
    hipMemcpy(matrix_aux, matrix_in, sizeof(double) * lengthX_in * lengthY_in, hipMemcpyDeviceToHost);

    std::cout << "Sizes: X = " << lengthX_in << "; Y = " << lengthY_in << "\n";
    std::cout.precision(precision);
    std::cout << std::scientific;
    for (unsigned i = 0u; i < lengthX_in; i++)
    {
        std::cout << "| ";
        for (unsigned j = 0u; j < lengthY_in; j++)
        {
            std::cout << matrix_aux[j * lengthX_in + i];
        }
        std::cout << " |\n";
    }
}

// Out-Placed Calculation Device

__device__ void _Add(double *vector_out, double *vectorLeft_in, double *vectorRight_in, unsigned length_in)
{
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < length_in)
    {
        vector_out[index] = vectorLeft_in[index] + vectorRight_in[index];
    }
}
__device__ void _Sub(double *vector_out, double *vectorLeft_in, double *vectorRight_in, unsigned length_in)
{
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < length_in)
    {
        vector_out[index] = vectorLeft_in[index] - vectorRight_in[index];
    }
}
__device__ void _Mul(double *vector_out, double *vectorLeft_in, double *vectorRight_in, unsigned length_in)
{
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < length_in)
    {
        vector_out[index] = vectorLeft_in[index] * vectorRight_in[index];
    }
}
__device__ void _Mul(double *vector_out, double *vectorLeft_in, double value_in, unsigned length_in)
{
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < length_in)
    {
        vector_out[index] = vectorLeft_in[index] * value_in;
    }
}

// Out-Placed Calculation Host
void MathGPU::Add(double *vector_out, double *vectorLeft_in, double *vectorRight_in, unsigned length_in, hipStream_t stream)
{
    unsigned length = MathGPU::properties->maxThreadsPerBlock;
    dim3 T(length);
    dim3 B((length_in + T.x - 1u) / T.x);
    _Add<<<T, B, 0, stream>>>(vector_out, vectorLeft_in, vectorRight_in, length_in);
}
void MathGPU::Sub(double *vector_out, double *vectorLeft_in, double *vectorRight_in, unsigned length_in, hipStream_t stream)
{
    unsigned length = MathGPU::properties->maxThreadsPerBlock;
    dim3 T(length);
    dim3 B((length_in + T.x - 1u) / T.x);
    _Sub<<<T, B, 0, stream>>>(vector_out, vectorLeft_in, vectorRight_in, length_in);
}
void MathGPU::Mul(double *vector_out, double *vectorLeft_in, double *vectorRight_in, unsigned length_in, hipStream_t stream)
{
    unsigned length = MathGPU::properties->maxThreadsPerBlock;
    dim3 T(length);
    dim3 B((length_in + T.x - 1u) / T.x);
    _Mul<<<T, B, 0, stream>>>(vector_out, vectorLeft_in, vectorRight_in, length_in);
}
void MathGPU::Mul(double *vector_out, double *vectorLeft_in, double value_in, unsigned length_in, hipStream_t stream)
{
    unsigned length = MathGPU::properties->maxThreadsPerBlock;
    dim3 T(length);
    dim3 B((length_in + T.x - 1u) / T.x);
    _Mul<<<T, B, 0, stream>>>(vector_out, vectorLeft_in, value_in, length_in);
}

// Out-Placed Calculation Device
__device__ void _Add(double *vector_inout, double *vector_in, unsigned length_in)
{
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < length_in)
    {
        vector_inout[index] += vector_in[index];
    }
}
__device__ void _Sub(double *vector_inout, double *vector_in, unsigned length_in)
{
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < length_in)
    {
        vector_inout[index] -= vector_in[index];
    }
}
__device__ void _Mul(double *vector_inout, double *vector_in, unsigned length_in)
{
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < length_in)
    {
        vector_inout[index] *= vector_in[index];
    }
}
__device__ void _Mul(double *vector_inout, double value_in, unsigned length_in)
{
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < length_in)
    {
        vector_inout[index] *= value_in;
    }
}

// In-Placed Calculation
void MathGPU::Add(double *vector_inout, double *vector_in, unsigned length_in, hipStream_t stream)
{
    unsigned length = MathGPU::properties->maxThreadsPerBlock;
    dim3 T(length);
    dim3 B((length_in + T.x - 1u) / T.x);
    _Add<<<T, B, 0, stream>>>(vector_inout, vector_in, length_in);
}
void MathGPU::Sub(double *vector_inout, double *vector_in, unsigned length_in, hipStream_t stream)
{
    unsigned length = MathGPU::properties->maxThreadsPerBlock;
    dim3 T(length);
    dim3 B((length_in + T.x - 1u) / T.x);
    _Sub<<<T, B, 0, stream>>>(vector_inout, vector_in, length_in);
}
void MathGPU::Mul(double *vector_inout, double *vector_in, unsigned length_in, hipStream_t stream)
{
    unsigned length = MathGPU::properties->maxThreadsPerBlock;
    dim3 T(length);
    dim3 B((length_in + T.x - 1u) / T.x);
    _Mul<<<T, B, 0, stream>>>(vector_inout, vector_in, length_in);
}
void MathGPU::Mul(double *vector_inout, double value_in, unsigned length_in, hipStream_t stream)
{
    unsigned length = MathGPU::properties->maxThreadsPerBlock;
    dim3 T(length);
    dim3 B((length_in + T.x - 1u) / T.x);
    _Mul<<<T, B, 0, stream>>>(vector_inout, value_in, length_in);
}

hipblasOperation_t MapStructure(MatrixStructure structure)
{
    hipblasOperation_t operation;
    switch (structure)
    {
    case MatrixStructure::Natural:
        operation = hipblasOperation_t::HIPBLAS_OP_N;
        break;
    case MatrixStructure::Transposed:
        operation = hipblasOperation_t::HIPBLAS_OP_T;
        break;
    default:
        std::cout << "Structure not defined for the left matrix. Using default structure.\n";
        return HIPBLAS_OP_N;
        break;
    }
    return operation;
}

// Matrix Multiplication
void MathGPU::MatrixMultiplication(double *matrix_out, double alpha, double beta,
                                   double *matrixLeft_in, MatrixStructure structureLeft, unsigned lengthLeftX_in, unsigned lengthLeftY_in,
                                   double *matrixRight_in, MatrixStructure structureRight, unsigned lengthRightX_in, unsigned lengthRightY_in,
                                   double *weight_in, hipStream_t stream)
{
    hipblasHandle_t handle;
    hipblasOperation_t left_OP, right_OP;
    hipblasCreate(&handle);
    hipblasSetStream(handle, stream);
    hipblasOperation_t left_OP = MapStructure(structureLeft);
    hipblasOperation_t right_OP = MapStructure(structureRight);
    if (lengthLeftY_in != lengthRightX_in)
    {
        std::cout << "Error: multiplication sizes do not match.\n";
        return;
    }
    hipblasDgemm(handle, left_OP, right_OP, lengthLeftX_in, lengthRightY_in, lengthLeftY_in, &alpha, matrixLeft_in, lengthLeftX_in, matrixRight_in, lengthRightX_in, &beta, matrix_out, lengthLeftX_in);
    hipblasDestroy(handle);
}

// Operators
void MathGPU::Operation(void (*operation_in)(double *matrix_out, double *matrixLeft_in, double *matrixRight_in, unsigned length_in),
                        double *matrix_out, double *matrixLeft_in, double *matrixRight_in, unsigned length_in,
                        unsigned strideOutput_in, unsigned strideLeft_in, unsigned strideRight_in, unsigned iteration_in, unsigned offsetOutput_in, unsigned offsetLeft_in, unsigned offsetRight_in)
{
    double *output = matrix_out + offsetOutput_in;
    double *left = matrixLeft_in + offsetLeft_in;
    double *right = matrixRight_in + offsetRight_in;
    if (length_in < strideLeft_in)
    {
        std::cout << "Warning: The behavior is undefined to output strides lower than the length of the operation.\n";
    }
    for (unsigned i = 0u; i < iteration_in; i++)
    {
        operation_in(output + i * strideOutput_in, left + i * strideLeft_in, right + i * strideRight_in, length_in);
    }
}
void MathGPU::Operation(void (*operation_in)(double *matrixLeft_inout, double *matrixRight_in, unsigned length_in),
                        double *matrixLeft_inout, double *matrixRight_in, unsigned length_in,
                        unsigned strideLeft_in, unsigned strideRight_in, unsigned iteration_in, unsigned offsetLeft_in, unsigned offsetRight_in)
{
    double *left = matrixLeft_inout + offsetLeft_in;
    double *right = matrixRight_in + offsetRight_in;
    if (length_in < strideLeft_in)
    {
        std::cout << "Warning: The behavior is undefined to left strides lower than the length of the operation.\n";
    }
    for (unsigned i = 0u; i < iteration_in; i++)
    {
        operation_in(left + i * strideLeft_in, right + i * strideRight_in, length_in);
    }
}

// Reducibles Operations
double MathGPU::Mean(double* value_out, double *vector_in, unsigned length_in, double *weight_in = NULL, bool transferResultCPU_in = true) {
    bool noWeight = weight_in == NULL;
    int stride = 1;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    if(noWeight){
        hipMalloc(&weight_in,sizeof(double));
        stride = 0;
        double value = 1.0/(double)length_in;
        hipMemcpy(weight_in,&value,sizeof(double),hipMemcpyKind::hipMemcpyHostToDevice);
    }
    hipblasDdot(handle,length_in,vector_in,1,weight_in,stride,value_out);
    double res = 0.0;
    if(transferResultCPU_in) {
        hipMemcpy(&res,value_out,1,hipMemcpyKind::hipMemcpyDeviceToHost);
    }
    if(noWeight){
        hipFree(weight_in);
    }
    return res;
    
}
void Mean(double *vector_out, double *matrix_in, unsigned lengthX_in, unsigned lengthY_in, double *weight_in = NULL){
    bool noWeight = weight_in == NULL;
    int stride = 1;
    int stride = 1;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    if(noWeight){
        hipMalloc(&weight_in,sizeof(double));
        stride = 0;
        double value = 1.0/(double)lengthY_in;
        hipMemcpy(weight_in,&value,sizeof(double),hipMemcpyKind::hipMemcpyHostToDevice);
    }
    hipblasDdot(handle,lengthY_in,matrix_in,lengthX_in,weight_in,stride,vector_out);
    double res = 0.0;
    if(noWeight){
        hipFree(weight_in);
    }
    return;
}
